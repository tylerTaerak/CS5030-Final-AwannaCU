
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>

#define MASK_SIZE 40401
#define SUB_WIDTH 201

__device__
void getNormalVisibility(
    int8_t sign,
    uint16_t x0,
    uint16_t y0,
    uint16_t x1,
    uint16_t y1,
    uint16_t observationElevation,
    short *data,
    char *visiblePoints,
    int16_t leftX,
    int16_t topY,
    uint16_t width
) {
    int tdx = threadIdx.x + blockIdx.x * blockDim.x;
    int tdy = threadIdx.y + blockIdx.y * blockDim.y;
    char finished = 0;
    int16_t deltaX = abs(x1 - x0);
    int16_t deltaY = y1 - y0;
    
    double maxSlope = -45;
    
    char incrementY = 1;
    
    if (deltaY < 0) {
        incrementY = -1;
        deltaY = -deltaY;
    }
    
    int error = 2 * deltaY - deltaX;
    
    uint16_t x = x0;
    uint16_t y = y0;
    
    while(!finished) {
        if (x == x1) {
            finished = 1;
        }
        
        double elevation = data[y * width + x] - observationElevation;
        double distance = sqrt((double)((x - x0) * (x - x0) + (y - y0) * (y - y0)));
        double slope = elevation / distance;
        
        if (slope > maxSlope) {
            visiblePoints[(y - topY) * SUB_WIDTH + (x - leftX)] = 1;
            maxSlope = slope;
        }
        
        if (error > 0) {
            y += incrementY;
            error -= 2 * deltaX;
        }
        
        error += 2 * deltaY;
        
        x += sign;
    }
}

__device__
void getInverseVisibility(
    int8_t sign,
    uint16_t x0,
    uint16_t y0,
    uint16_t x1,
    uint16_t y1,
    uint16_t observationElevation,
    short *data,
    char *visiblePoints,
    int16_t leftX,
    int16_t topY,
    uint16_t width
) {
    int tdx = threadIdx.x + blockIdx.x * blockDim.x;
    int tdy = threadIdx.y + blockIdx.y * blockDim.y;
    char finished = 0;
    int16_t deltaX = x1 - x0;
    int16_t deltaY = abs(y1 - y0);
    
    double maxSlope = -45;
    
    char incrementX = 1;
    
    if (deltaX < 0) {
        incrementX = -1;
        deltaX = -deltaX;
    }
    
    int error = 2 * deltaX - deltaY;
    
    uint16_t x = x0;
    uint16_t y = y0;
    
    while (!finished) {
        if (y == y1) {
            finished = 1;
        }
        
        double elevation = data[y * width + x] - observationElevation;
        double distance = sqrt((double)((x - x0) * (x - x0) + (y - y0) * (y - y0)));
        double slope = elevation / distance;
        
        if (slope > maxSlope) {
            visiblePoints[(y - topY) * SUB_WIDTH + (x - leftX)] = 1;
            maxSlope = slope;
        }
        
        if (error > 0) {
            x += incrementX;
            error -= 2 * deltaY;
        }
        
        error += 2 * deltaX;
        
        y += sign;
    }
}

__device__
void getVisibility(
    uint16_t x0,
    uint16_t y0,
    uint16_t x1,
    uint16_t y1,
    short *data,
    char *visiblePoints,
    int16_t leftX,
    int16_t topY,
    uint16_t width
) {
    uint16_t observationElevation = data[y0 * width + x0];
    
    if (abs(y1 - y0) < abs(x1 - x0)) {
        if (x0 > x1) {
            getNormalVisibility(-1, x0, y0, x1, y1, observationElevation, data, visiblePoints, leftX, topY, width);
        }
        else {
            getNormalVisibility(1, x0, y0, x1, y1, observationElevation, data, visiblePoints, leftX, topY, width);
        }
    }
    else {
        if (y0 > y1) {
            getInverseVisibility(-1, x0, y0, x1, y1, observationElevation, data, visiblePoints, leftX, topY, width);
        }
        else {
            getInverseVisibility(1, x0, y0, x1, y1, observationElevation, data, visiblePoints, leftX, topY, width);
        }
    }
}

__device__
uint32_t getVisibilityInAreaOfInterest(
    uint16_t x0,
    uint16_t y0,
    uint8_t radius,
    short *data,
    char *visiblePoints,
    uint16_t width,
    uint16_t height
) {
    uint32_t totalVisiblePoints = 0;
    
    int16_t leftX   = (x0 - radius) < 0 ? 0 : (x0 - radius) ;
    int16_t topY    = (y0 - radius) < 0 ? 0 : (y0 - radius) ;
    int16_t rightX  = (x0 + radius) >= width  ? width - 1  : (x0 + radius) ;
    int16_t bottomY = (y0 + radius) >= height ? height - 1 : (y0 + radius) ;
    
    uint16_t x = leftX;
    uint16_t y = topY;
    
    for (; x < rightX; x++) {
        getVisibility(x0, y0, x, y, data, visiblePoints, leftX, topY, width);
    }
    getVisibility(x0, y0, x, y, data, visiblePoints, leftX, topY, width);
    
    for (y += 1; y < bottomY; y++) {
        getVisibility(x0, y0, x, y, data, visiblePoints, leftX, topY, width);
    }
    getVisibility(x0, y0, x, y, data, visiblePoints, leftX, topY, width);
    
    for (x -= 1; x > leftX; x--) {
        getVisibility(x0, y0, x, y, data, visiblePoints, leftX, topY, width);
    }
    getVisibility(x0, y0, x, y, data, visiblePoints, leftX, topY, width);
    
    for (y -= 1; y > topY; y--) {
        getVisibility(x0, y0, x, y, data, visiblePoints, leftX, topY, width);
    }
    
    for (int i = 0; i < (SUB_WIDTH * SUB_WIDTH); i++) {
        totalVisiblePoints += visiblePoints[i];
    }
    
    return totalVisiblePoints;
}

__global__
void calcViewshed(short *data, uint32_t *viewshed, uint8_t radius, uint16_t width, uint16_t height) {
    char visiblePoints[MASK_SIZE];
    
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    for (int i = 0; i < MASK_SIZE; i++) {
        visiblePoints[i] = 0;
    }
    
    if (x < width && y < height) {
        viewshed[y * width + x] = getVisibilityInAreaOfInterest(x, y, radius, data, visiblePoints, width, height);
    }
}

/* This function is utilized by the distributed GPU implementation */
extern "C" void startKernel(short *data, uint32_t *viewshed, uint8_t radius, uint16_t width, uint16_t height)
{
  short *data_d;
  uint32_t *viewshed_d;

  hipMalloc((void**) &data_d, width * height * sizeof(short));
  hipMalloc((void**) &viewshed_d, width * height * sizeof(uint32_t));

  hipMemcpy(data_d, data, width * height * sizeof(short), hipMemcpyHostToDevice);

  /* dimensions used to calculate a 1000 x 1000 space */
  dim3 DimGrid(100, 100, 1); 
  dim3 DimBlock(10, 10, 1);

  calcViewshed<<<DimGrid, DimBlock>>>(data_d, viewshed_d, radius, width, height);

  hipMemcpy(viewshed, viewshed_d, width * height * sizeof(uint32_t), hipMemcpyDeviceToHost);
}


int main() {
    short *data_h;
    uint32_t *viewshed_h;
    short *data_d;
    uint32_t *viewshed_d;
    
    // cudaEvent_t startEvent, stopEvent;    // CUDA events used to compute the elapsed time of the kernal functions.
    // cudaEventCreate(&startEvent);
    // cudaEventCreate(&stopEvent);
    
    // float elapsedTime = 0;                // Actual elapsed time of the kernal functions.
    
    uint8_t radius = 100;
    
    uint32_t size;
    
    uint16_t width = 6000;
    uint16_t height = 6000;
    
    // FILE* f = fopen("test.raw", "rb");
    FILE* f = fopen("srtm_14_04_6000x6000_short16.raw", "rb");
    
    if (f == NULL) {
        printf("Error: input file srtm_14_04_6000x6000_short16.raw could not be opened.\n");
        return 1;
    }
    
    fseek(f, 0, SEEK_END);
    size = ftell(f) / sizeof(short);
    fseek(f, 0, SEEK_SET);
    
    hipMalloc((void**) &data_d, size * sizeof(short));
    hipMalloc((void**) &viewshed_d, size * sizeof(uint32_t));
    
    data_h = (short*) malloc(sizeof(short) * size);
    
    fread(data_h, sizeof(short), size, f);
    
    fclose(f);
    
    hipMemcpy(data_d, data_h, size * sizeof(short), hipMemcpyHostToDevice);
    
    dim3 DimGrid(375,375,1);
    dim3 DimBlock(16,16,1);
    
    calcViewshed<<<DimGrid, DimBlock>>>(data_d, viewshed_d, radius, width, height);
    
    viewshed_h = (uint32_t*) malloc(sizeof(uint32_t) * size);
    
    hipMemcpy(viewshed_h, viewshed_d, size * sizeof(uint32_t), hipMemcpyDeviceToHost);
    
    f = fopen("srtm_14_04_6000x6000_int32_gpu_100.raw", "wb");
    
    fwrite(viewshed_h, sizeof(uint32_t), size, f);
    
    fclose(f);
    
    free(data_h);
    free(viewshed_h);
    
    hipFree(data_d);
    hipFree(viewshed_d);
    
    return 0;
}
